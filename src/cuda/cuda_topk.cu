// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

// #include "cuda_topk_bitonic_sort_v0.cuh"
// #include "cuda_topk_bitonic_sort_v3.cuh"
// #include "cuda_topk_bitonic_sort_v4.cuh"
// #include "cuda_topk_bitonic_sort_v6.cuh"
// #include "cuda_topk_bitonic_sort_v7.cuh"
// #include "cuda_topk_bitonic_sort_v8.cuh"
// #include "cuda_topk_bitonic_sort_v9.cuh"
#include "cuda_topk_bitonic_sort_v10.cuh"
// #include "cuda_topk_bitonic_sort_v11.cuh"
// #include "cuda_topk_bitonic_sort_v12.cuh"
// #include "cuda_topk_bitonic_sort_v13.cuh"
// #include "cuda_topk_bitonic_sort_v17.cuh"
#include "cuda_topk_bitonic_sort_v19.cuh"
// #include "cuda_topk_bitonic_sort_v21.cuh"
// #include "cuda_topk_bitonic_sort_v22.cuh"
#include "cuda_topk_bitonic_sort_v23.cuh"
#include "cuda_topk_bitonic_sort_v24.cuh"

namespace Generators {
namespace cuda {

const char* GetBitonicBaselineDescription() {
  return bitonic_v24::kAlgoDescription;
}
void RunTopKViaHybridSort(SamplingData* data, hipStream_t stream, float* scores_in, float* scores_out, int* indices_out, int vocab_size, int batch_size, int k, float temperature, int num_partitions, int sort_size) {
  bitonic_v24::RunTopKViaMapReduceBitonicSort(data, stream, scores_in, scores_out, indices_out, vocab_size, batch_size, k, temperature, num_partitions, sort_size);
}

const char* GetBitonicTreatmentDescription(){
  return bitonic_v19::kAlgoDescription;
}
void RunTopKViaMapReduceBitonicSort(SamplingData* data, hipStream_t stream, float* scores_in, float* scores_out, int* indices_out, int vocab_size, int batch_size, int k, float temperature, int num_partitions, int sort_size) {
  bitonic_v19::RunTopKViaMapReduceBitonicSort(data, stream, scores_in, scores_out, indices_out, vocab_size, batch_size, k, temperature, num_partitions, sort_size);
}

int GetBestSortSize(int vocab_size, int batch_size, int k) {
      if (k <= 4)
          return 0;

      if (vocal_size > 256 * 1024)
        return 8192;

      if (vocab_size >= 147456)
          return 4096;
      else if (vocab_size < 49152) {
        if (k <= 8)
          return 0;
      }
      else {
        if (k < 8)
          return 0;
        if (vocab_size >= 65536 || batch_size >= 4)
            return 2048;
      }
          
      return 1024;
}

void GetTopKSubset(SamplingData* data, hipStream_t stream, float* scores_in, float* scores_out, int* indices_out, int vocab_size, int batch_size, int k, float temperature) {
  TopKConfig chosen_config;
  if (k <= 8) {
    chosen_config.algorithm = TopKAlgorithm::SELECTION_SORT;
  } else if (k <= 64) {
    chosen_config = BenchmarkAndGetBestAlgorithm(data, stream, vocab_size, batch_size, k);
  }

  switch (chosen_config.algorithm) {
    case TopKAlgorithm::SELECTION_SORT:
      RunTopKViaSelectionSort(data, stream, scores_in, scores_out, indices_out, vocab_size, batch_size, k, temperature);
      break;
    case TopKAlgorithm::BITONIC_SORT:
      RunTopKViaMapReduceBitonicSort(data, stream, scores_in, scores_out, indices_out, vocab_size, batch_size, k, temperature, chosen_config.num_partitions, chosen_config.sort_size);
      break;
    default:
      RunTopKViaFullSort(data, stream, scores_in, scores_out, indices_out, vocab_size, batch_size, k, temperature);
      break;
  }
}

}  // namespace cuda
}  // namespace Generators

