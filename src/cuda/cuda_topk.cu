#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <cub/device/device_segmented_radix_sort.cuh>

#include "cuda_topk.h"
#include "cuda_topk_full_sort.cuh"
#include "cuda_topk_hybrid_sort.cuh"
#include "cuda_topk_select_sort.cuh"

namespace Generators {
namespace cuda {

// Helper to determine the optimal partition size for the hybrid sort algorithm
// based on vocabulary and batch size.
inline int GetHybridSortPartitionSize(int vocab_size, int batch_size) {
  if (vocab_size >= 147456) {
    return (vocab_size > 256 * 1024) ? 8192 : 4096;
  } else {
    if (vocab_size >= 65536 || batch_size >= 4 && vocab_size > 49152) {
      return 2048;
    }
  }

  return 1024;
}

TopkData::TopkData(int batch_size, int vocab_size, hipStream_t stream) {
  // The intermediate buffers are used by hybrid and full sort algorithms.
  int partition_size = GetHybridSortPartitionSize(vocab_size, batch_size);
  size_t intermediate_buffer_elements = GetHybridSortIntermediateSize(batch_size, vocab_size, partition_size);

  size_t vocab_batch_size = static_cast<size_t>(vocab_size) * batch_size;

  // Selection sort uses a buffer of batch_size * 64 elements, which is smaller than intermediate_buffer_elements.
  size_t max_buffer_elements = std::max(vocab_batch_size, intermediate_buffer_elements);

  // Allocate all necessary device memory
  intermediate_indices_1 = CudaMallocArray<int>(max_buffer_elements);
  intermediate_indices_2 = CudaMallocArray<int>(max_buffer_elements);
  intermediate_scores_1 = CudaMallocArray<float>(max_buffer_elements);
  intermediate_scores_2 = CudaMallocArray<float>(max_buffer_elements);
  batch_offsets = CudaMallocArray<int>(batch_size + 1);

  cub_temp_storage_bytes = GetFullSortCubTempStorageBytes(vocab_batch_size, batch_size, stream);
  cub_temp_storage = CudaMallocArray<unsigned char>(this->cub_temp_storage_bytes);
}

void TopkDataCompact::CompactOutput(int batch_size, int vocab_size, hipStream_t stream, int k) {
  topk_scores_compact = CudaMallocArray<float>(static_cast<size_t>(batch_size) * k);
  topk_indices_compact = CudaMallocArray<int>(static_cast<size_t>(batch_size) * k);
  dim3 grid(batch_size);
  dim3 block(256);
  CompactStridedData<float><<<grid, block, 0, stream>>>(topk_scores, topk_scores_compact.get(), k, batch_size, topk_stride);
  CompactStridedData<int><<<grid, block, 0, stream>>>(topk_indices, topk_indices_compact.get(), k, batch_size, topk_stride);
  hipStreamSynchronize(stream);
}

void GetTopK(TopkData* topk_data, hipStream_t stream, const float* scores_in, int vocab_size, int batch_size, int k) {
  assert(topk_data != nullptr);

  if (k > kHybridSortMaxK) {
    RunTopKViaFullSort(topk_data, stream, scores_in, vocab_size, batch_size, k);
  } else if (k <= 8 || vocab_size < 1024) {
    // NOTE: This modifies scores_in in-place
    RunTopKViaSelectionSort(topk_data, stream, scores_in, vocab_size, batch_size, k);
  } else {
    int partition_size = GetHybridSortPartitionSize(vocab_size, batch_size);
    RunTopKViaHybridSort(topk_data, stream, scores_in, vocab_size, batch_size, k, partition_size);
  }
}

}  // namespace cuda
}  // namespace Generators
