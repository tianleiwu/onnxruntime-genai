#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <hip/hip_runtime.h>
#include <math.h>

#include <cfloat>
#include <cstdio>
#include <cstdlib>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <limits>

#include "cuda_sampling.h"
#include "cuda_topk.h"
#include "smartptrs.h"
#include "span.h"

namespace Generators {
namespace cuda {

constexpr int kMaxThreads = 1024;
constexpr int kGPUWarpSize = 32;

// Softmax Kernels and Launchers

template <typename T, typename AccumT>
struct MaxFloat {
  __device__ __forceinline__ AccumT operator()(AccumT max, T v) const {
    return ::max(max, (AccumT)v);
  }
};

template <typename T>
struct Max {
  __device__ __forceinline__ T operator()(T a, T b) const {
    return a < b ? b : a;
  }
};

template <typename T, typename AccumT>
struct SumExpFloat {
  __device__ __forceinline__ SumExpFloat(AccumT v)
      : max_k(v) {}

  __device__ __forceinline__ AccumT operator()(AccumT sum, T v) const {
    return sum + exp((AccumT)v - max_k);
  }

  const AccumT max_k;
};

template <typename T>
struct Add {
  __device__ __forceinline__ T operator()(T a, T b) const {
    return a + b;
  }
};

// aligned vector generates vectorized load/store on CUDA
template <typename T, int vec_size>
struct alignas(sizeof(T) * vec_size) aligned_vector {
  T val[vec_size];
};

template <template <typename, typename> class Reduction, int ILP, typename T, typename AccumT>
__device__ __forceinline__ AccumT IlpReduce(int shift, T* data, int size, const Reduction<T, AccumT>& r, AccumT defaultVal) {
  using LoadT = aligned_vector<T, ILP>;
  AccumT threadVal = defaultVal;
  int offset = threadIdx.x;
  // shift and do 1
  if (shift > 0) {
    data -= shift;
    size += shift;
    if (threadIdx.x >= shift && threadIdx.x < size) {
      threadVal = r(threadVal, data[offset]);
    }
    size -= blockDim.x;
    data += blockDim.x;
  }
  if (size <= 0) return threadVal;
  int last = size % (ILP * blockDim.x);
  T v[ILP];
  LoadT* value = reinterpret_cast<LoadT*>(&v);
  for (; offset * ILP < (size - last); offset += blockDim.x) {
    *value = reinterpret_cast<LoadT*>(data)[offset];
#pragma unroll
    for (int j = 0; j < ILP; ++j) {
      threadVal = r(threadVal, v[j]);
    }
  }
  offset = size - last + threadIdx.x;
  // Epilogue
  for (; offset < size; offset += blockDim.x)
    threadVal = r(threadVal, data[offset]);
  return threadVal;
}

template <template <typename> class Reduction, typename AccumT>
__device__ __forceinline__ AccumT SoftmaxReduce(AccumT* smem, AccumT val, const Reduction<AccumT>& r, AccumT defaultVal) {
  // To avoid RaW races from chaining SoftmaxReduce calls together, we need a sync here
  __syncthreads();
  smem[threadIdx.x] = val;
  __syncthreads();
  AccumT warpVal = defaultVal;
  // First warp will perform per-warp reductions for the remaining warps
  if (threadIdx.x < kGPUWarpSize) {
    int warps_per_block = blockDim.x / kGPUWarpSize;
    for (int i = 0; i < warps_per_block; ++i) {
      warpVal = r(warpVal, smem[i * kGPUWarpSize + threadIdx.x]);
    }
    smem[threadIdx.x] = warpVal;
  }
  __syncthreads();
  // First thread will perform a reduction of the above per-warp reductions
  AccumT blockVal = defaultVal;
  if (threadIdx.x == 0) {
#pragma unroll
    for (int i = 0; i < kGPUWarpSize; ++i) {
      blockVal = r(blockVal, smem[i]);
    }
    smem[0] = blockVal;
  }
  // Sync and broadcast
  __syncthreads();
  return smem[0];
}

dim3 SoftmaxGetBlockSize(int ILP, uint64_t size) {
  uint64_t block_size = 1;
  uint64_t max_block_size = min(size / ILP, static_cast<uint64_t>(kMaxThreads));
  // In the vectorized case we want to trade off allowing more of the buffers to be accessed
  // in a vectorized way against wanting a larger block size to get better utilisation.
  // In general with ILP you can have (ILP-1)/ILP of the buffer accessed vectorised, at the risk
  // of having a very small block size. We choose to keep >= 1/2 of the buffer vectorised while
  // allowing a larger block size.
  if (ILP > 1) {
    max_block_size /= 2;
  }
  while (block_size < max_block_size) block_size *= 2;
  // Launch at least a single warp - the kernel assumes that.
  block_size = max(block_size, static_cast<uint64_t>(kGPUWarpSize));
  return dim3(static_cast<unsigned int>(block_size));
}

template <typename T, typename AccumT, typename OutT>
struct LogSoftmaxForwardEpilogue {
  __device__ __forceinline__ LogSoftmaxForwardEpilogue(AccumT max_input, AccumT sum)
      : max_input(max_input), logsum(log(sum)) {}

  __device__ __forceinline__ OutT operator()(T input) const {
    return static_cast<OutT>((AccumT)input - max_input - logsum);
  }

  const AccumT max_input;
  const AccumT logsum;
};

template <typename T, typename AccumT, typename OutT>
struct SoftmaxForwardEpilogue {
  __device__ __forceinline__ SoftmaxForwardEpilogue(AccumT max_input, AccumT sum)
      : max_input(max_input), sum(sum) {}

  __device__ __forceinline__ OutT operator()(T input) const {
    return static_cast<OutT>(exp((AccumT)input - max_input) / sum);
  }

  const AccumT max_input;
  const AccumT sum;
};

/**
 * This will apply the Epilogue with vectorized reads & writes when input & output have the same shift
 */
template <int ILP, typename scalar_t, typename accum_t, typename outscalar_t, template <typename, typename, typename> class Epilogue>
__device__ __forceinline__ void WriteFpropResultsVectorized(int size,
                                                            const int shift,
                                                            scalar_t* input,
                                                            outscalar_t* output,
                                                            Epilogue<scalar_t, accum_t, outscalar_t> epilogue) {
  using LoadT = aligned_vector<scalar_t, ILP>;
  using StoreT = aligned_vector<outscalar_t, ILP>;
  int offset = threadIdx.x;
  // if unaligned, do one value / thread and move on, guaranteeing aligned reads/writes later
  if (shift > 0) {
    input -= shift;
    output -= shift;
    size += shift;
    if (threadIdx.x >= shift && threadIdx.x < size) {
      output[offset] = epilogue(input[offset]);
    }
    size -= blockDim.x;
    input += blockDim.x;
    output += blockDim.x;
  }
  if (size <= 0) return;
  const int last = size % (ILP * blockDim.x);
  scalar_t in_v[ILP];
  LoadT* in_value = reinterpret_cast<LoadT*>(&in_v);
  outscalar_t out_v[ILP];
  StoreT* out_value = reinterpret_cast<StoreT*>(&out_v);
  for (; offset * ILP < (size - last); offset += blockDim.x) {
    *in_value = reinterpret_cast<LoadT*>(input)[offset];
#pragma unroll
    for (int j = 0; j < ILP; ++j) {
      out_v[j] = epilogue(in_v[j]);
    }
    reinterpret_cast<StoreT*>(output)[offset] = *out_value;
  }
  offset = size - last + threadIdx.x;
  // handle the tail
  for (; offset < size; offset += blockDim.x) {
    output[offset] = epilogue(input[offset]);
  }
}

/**
 * This will apply the Epilogue with non-vectrorized reads & writes for the general case
 */
template <int ILP, typename scalar_t, typename accum_t, typename outscalar_t, template <typename, typename, typename> class Epilogue>
__device__ __forceinline__ void WriteFpropResults(int classes,
                                                  scalar_t* input,
                                                  outscalar_t* output,
                                                  Epilogue<scalar_t, accum_t, outscalar_t> epilogue) {
  int offset = threadIdx.x;
  int last = classes % (ILP * blockDim.x);
  // Main bulk of loop with ILP
  for (; offset < classes - last; offset += blockDim.x * ILP) {
    scalar_t tmp[ILP];
#pragma unroll
    for (int j = 0; j < ILP; ++j) {
      tmp[j] = input[offset + j * blockDim.x];
    }
#pragma unroll
    for (int j = 0; j < ILP; ++j) {
      output[offset + j * blockDim.x] = epilogue(tmp[j]);
    }
  }
  // Remainder - no ILP
  for (; offset < classes; offset += blockDim.x) {
    output[offset] = epilogue(input[offset]);
  }
}

template <int ILP, typename scalar_t, typename accscalar_t, typename outscalar_t,
          template <typename, typename, typename> class Epilogue>
__global__ void SoftmaxBlockForward(outscalar_t* output, scalar_t* input, int classes,
                                    int input_stride, int output_stride) {
  extern __shared__ unsigned char smem[];
  auto sdata = reinterpret_cast<accscalar_t*>(smem);
  // forward pointers to batch[blockIdx.x]
  // each block handles a sample in the mini-batch
  input += blockIdx.x * input_stride;
  output += blockIdx.x * output_stride;
  const int input_align_bytes = ILP * sizeof(scalar_t);
  const int output_align_bytes = ILP * sizeof(outscalar_t);
  const int shift = ((uint64_t)input) % input_align_bytes / sizeof(scalar_t);
  const int output_shift = ((uint64_t)output) % output_align_bytes / sizeof(outscalar_t);
  // find the max
  accscalar_t threadMax = IlpReduce<MaxFloat, ILP, scalar_t, accscalar_t>(
      shift, input, classes, MaxFloat<scalar_t, accscalar_t>(), -std::numeric_limits<accscalar_t>::max());
  accscalar_t max_k = SoftmaxReduce<Max, accscalar_t>(
      sdata, threadMax, Max<accscalar_t>(), -std::numeric_limits<accscalar_t>::max());
  // reduce all values
  accscalar_t threadExp = IlpReduce<SumExpFloat, ILP, scalar_t, accscalar_t>(
      shift, input, classes, SumExpFloat<scalar_t, accscalar_t>(max_k), static_cast<accscalar_t>(0));
  accscalar_t sumAll = SoftmaxReduce<Add, accscalar_t>(
      sdata, threadExp, Add<accscalar_t>(), static_cast<accscalar_t>(0));
  Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(max_k, sumAll);
  if (shift == output_shift) {
    WriteFpropResultsVectorized<ILP, scalar_t, accscalar_t, outscalar_t, Epilogue>(classes, shift, input, output, epilogue);
  } else {
    WriteFpropResults<ILP, scalar_t, accscalar_t, outscalar_t, Epilogue>(classes, input, output, epilogue);
  }
}

template <bool is_log_softmax>
void DispatchBlockwiseSoftmaxForward(hipStream_t stream, float* output, const float* input, int softmax_elements,
                                     int input_stride, int output_stride, int batch_count) {
  dim3 grid(batch_count);
  constexpr int ILP = sizeof(float4) / sizeof(float);
  dim3 block = SoftmaxGetBlockSize(ILP, softmax_elements);
  if (is_log_softmax) {
    SoftmaxBlockForward<ILP, float, float, float, LogSoftmaxForwardEpilogue>
        <<<grid, block, block.x * sizeof(float), stream>>>(output, const_cast<float*>(input),
                                                           softmax_elements, input_stride, output_stride);
  } else {
    SoftmaxBlockForward<ILP, float, float, float, SoftmaxForwardEpilogue>
        <<<grid, block, block.x * sizeof(float), stream>>>(output, const_cast<float*>(input),
                                                           softmax_elements, input_stride, output_stride);
  }
  CUDA_CHECK(hipGetLastError());
}
template void DispatchBlockwiseSoftmaxForward<true>(hipStream_t, float*, const float*, int, int, int, int);
template void DispatchBlockwiseSoftmaxForward<false>(hipStream_t, float*, const float*, int, int, int, int);

}  // namespace cuda
}  // namespace Generators
