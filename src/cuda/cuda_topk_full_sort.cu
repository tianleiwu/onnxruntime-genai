#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "cuda_topk_helper.h"

namespace Generators {
namespace cuda {

__global__ void PopulateIndices(int* indices, int size, int batch_size) {
  int global_index = threadIdx.x + blockIdx.x * blockDim.x;
  int index = global_index % size;
  if (global_index < size * batch_size) {
    indices[global_index] = index;
  }
}

void LaunchPopulateIndices(int* indices, int size, int batch_size, hipStream_t stream) {
  dim3 grid((batch_size * size / 256) + 1, 1, 1);
  dim3 block(256, 1, 1);
  PopulateIndices<<<grid, block, 0, stream>>>(indices, size, batch_size);
  CUDA_CHECK(hipGetLastError());
}

__global__ void PopulateOffsets(int* offsets, int size, int batch_size) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < batch_size + 1)
    offsets[index] = index * size;
}

void LaunchPopulateOffsets(int* offsets, int size, int batch_size, hipStream_t stream) {
  dim3 grid(int(batch_size / 128) + 1, 1, 1);
  dim3 block(128, 1, 1);
  PopulateOffsets<<<grid, block, 0, stream>>>(offsets, size, batch_size);
  CUDA_CHECK(hipGetLastError());
}

// Sorting Kernel Launcher

template <typename T>
void LaunchSortPairs(void* d_temp_storage,
                     size_t temp_storage_bytes,
                     const T* d_keys_in,
                     T* d_keys_out,
                     const int* d_values_in,
                     int* d_values_out,
                     int num_items,
                     int num_segments,
                     int* d_offsets,
                     hipStream_t stream,
                     bool is_descending) {
  if (is_descending) {
    CUDA_CHECK(hipcub::DeviceSegmentedRadixSort::SortPairsDescending(d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out,
                                                                  d_values_in, d_values_out, num_items, num_segments, d_offsets, d_offsets + 1, 0, sizeof(T) * 8, stream));
  } else {
    CUDA_CHECK(hipcub::DeviceSegmentedRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out,
                                                        d_values_in, d_values_out, num_items, num_segments, d_offsets, d_offsets + 1, 0, sizeof(T) * 8, stream));
  }
}

void LaunchSort(SamplingData* data, hipStream_t stream, float* scores_in, float* scores_out, int* indices_out, int vocab_size, int batch_size) {
  // Sort indices and scores
  LaunchPopulateOffsets(data->offsets.get(), vocab_size, batch_size, stream);
  LaunchPopulateIndices(data->indices_in.get(), vocab_size, batch_size, stream);
  LaunchSortPairs<float>(data->temp_buffer.get(), data->temp_storage_bytes, scores_in, scores_out,
                         data->indices_in.get(), indices_out, vocab_size * batch_size, batch_size, data->offsets.get(),
                         stream, /*is_descending*/ true);
}

void RunTopKViaFullSort(SamplingData* data, hipStream_t stream, float* scores_in, float* scores_out, int* indices_out, int vocab_size, int batch_size, int k, float temperature) {
  // Step 1: Perform a full, segmented sort on the input scores.
  float* sorted_scores = data->scores_buffer.get();
  int* sorted_indices = data->indices_in.get();
  LaunchSort(data, stream, scores_in, sorted_scores, sorted_indices, vocab_size, batch_size);

  // Step 2: Launch a specialized kernel that leverages the pre-sorted nature of the data.
  // This is faster than the general-purpose `ApplySoftmaxToTopK` because it avoids a reduction.
  ApplySoftmaxToSortedTopK<true>(stream, scores_out, indices_out, sorted_scores, sorted_indices, k, batch_size, vocab_size, temperature);
}

}
}
