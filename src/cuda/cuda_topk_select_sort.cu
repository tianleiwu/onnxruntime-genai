#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "cuda_topk_helper.h"
#include "cuda_sampling.h"

namespace Generators {
namespace cuda {

// START of improved Top-K kernel (Selection Sort approach)
struct TopK_2 {
  int p = INT_MAX;
  float u = -FLT_MAX;

  __device__ __forceinline__ void insert(float elem, int elem_id) {
    if (elem > u || (elem == u && elem_id < p)) {
      u = elem;
      p = elem_id;
    }
  }

  __device__ __forceinline__ void init() {
    u = -FLT_MAX;
    p = -1;
  }
};

__device__ __forceinline__ TopK_2 reduce_topk_op_2(TopK_2 const& a, TopK_2 const& b) {
  return a.u > b.u ? a : (a.u == b.u && a.p < b.p) ? a
                                                   : b;
}

template <int kBlockSize>
__global__ void GetTopKKernel(int* indices_out, float* scores_in, float* scores_out, int batch_size, int vocab_size, int k, float temperature) {
  int batch = blockIdx.x;
  int tid = threadIdx.x;
  TopK_2 partial;

  float const MAX_T_VAL = FLT_MAX;

  for (int ite = 0; ite < k; ite++) {
    partial.init();
    for (auto elemId = tid; elemId < vocab_size; elemId += kBlockSize) {
      float elem = scores_in[elemId + batch * vocab_size];
      partial.insert(elem, elemId);
    }
    // reduce in thread block
    typedef hipcub::BlockReduce<TopK_2, kBlockSize> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    TopK_2 top_k_sequence = BlockReduce(temp_storage).Reduce(partial, reduce_topk_op_2);

    if (tid == 0) {
      scores_out[ite + batch * k] = top_k_sequence.u / temperature;
      indices_out[ite + batch * k] = top_k_sequence.p;

      // set the max value to -MAX_T_VAL so that the value doesn't get picked again
      scores_in[batch * vocab_size + top_k_sequence.p] = -MAX_T_VAL;
    }

    // Attention: Add a block-level memory fence here.
    // This ensures that the write to global memory by thread 0 is visible
    // to all other threads in the block before the next iteration begins.
    // Without this, other threads might read the old (pre-modification)
    // score in the next iteration, leading to the same item being picked again.
    __threadfence_block();

    __syncthreads();
  }
}

// Launcher for the improved Top-K kernel.
void LaunchImprovedGetTopK(hipStream_t stream, float* scores_in, float* scores_out, int* indices_out, int vocab_size, int batch_size, int k, float temperature) {
  dim3 grid(batch_size, 1, 1);
  // Use a larger block size for better hardware utilization, as in the improved file.
  dim3 block(1024, 1, 1);
  GetTopKKernel<1024><<<grid, block, 0, stream>>>(indices_out, scores_in, scores_out, batch_size, vocab_size, k, temperature);
  CUDA_CHECK(hipGetLastError());
}

void RunTopKViaSelectionSort(SamplingData* data, hipStream_t stream, float* scores_in, float* scores_out, int* indices_out, int vocab_size, int batch_size, int k, float temperature) {
  // The output of the kernel will be the temperature-scaled scores. We'll store
  // these in another pre-allocated buffer, `scores_buffer`.
  float* scaled_scores = data->scores_buffer.get();

  // Attention: The kernel modifies the `scores_in` tensor in-place.
  // This might have unintended side effects on the original `scores_in` tensor if it is used elsewhere later.
  LaunchImprovedGetTopK(stream, scores_in, scaled_scores, indices_out, vocab_size, batch_size, k, temperature);

  // Finally, apply softmax to the scaled scores to get the final probabilities,
  // writing the result to the `scores_out` buffer.
  DispatchBlockwiseSoftmaxForward<false>(stream, scores_out, const_cast<const float*>(scaled_scores), k, k, k, batch_size);
}

}  // namespace cuda
}  // namespace Generators
