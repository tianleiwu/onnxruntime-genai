#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <hip/hip_runtime.h>
#include <math.h>

#include <cfloat>
#include <cstdio>
#include <cstdlib>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <limits>

#include "cuda_sampling.h"
#include "cuda_topk.h"
#include "smartptrs.h"
#include "span.h"

namespace Generators {
namespace cuda {

constexpr int kMaxThreads = 1024;
constexpr int kGPUWarpSize = 32;

__global__ void InitCurandStates(unsigned long long seed, hiprandState* states, int batch_size) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index >= batch_size)
    return;

  hiprand_init(seed, index, 0, &states[index]);
}

SamplingData::SamplingData(unsigned long long random_seed, int batch_size, int vocab_size, hipStream_t stream) : TopkData(batch_size, vocab_size, stream) {
  const size_t vocab_batch_size = static_cast<size_t>(vocab_size) * batch_size;

  prefix_sums = CudaMallocArray<float>(vocab_batch_size);
  scores_adjusted = CudaMallocArray<float>(vocab_batch_size);
  prefix_sums_adjusted = CudaMallocArray<float>(vocab_batch_size);

  thresholds = CudaMallocArray<float>(batch_size);
  curand_states = CudaMallocArray<hiprandState>(batch_size);

  InitCurandStates<<<CeilDiv(batch_size, 128), 128, 0, stream>>>(random_seed, curand_states.get(), batch_size);
  CUDA_CHECK(hipGetLastError());
}

// Softmax Kernels and Launchers

template <typename T, typename AccumT>
struct MaxFloat {
  __device__ __forceinline__ AccumT operator()(AccumT max, T v) const {
    return ::max(max, (AccumT)v);
  }
};

template <typename T>
struct Max {
  __device__ __forceinline__ T operator()(T a, T b) const {
    return a < b ? b : a;
  }
};

template <typename T, typename AccumT>
struct SumExpFloat {
  __device__ __forceinline__ SumExpFloat(AccumT v)
      : max_k(v) {}

  __device__ __forceinline__ AccumT operator()(AccumT sum, T v) const {
    return sum + exp((AccumT)v - max_k);
  }

  const AccumT max_k;
};

template <typename T>
struct Add {
  __device__ __forceinline__ T operator()(T a, T b) const {
    return a + b;
  }
};

// aligned vector generates vectorized load/store on CUDA
template <typename T, int vec_size>
struct alignas(sizeof(T) * vec_size) aligned_vector {
  T val[vec_size];
};

template <template <typename, typename> class Reduction, int ILP, typename T, typename AccumT>
__device__ __forceinline__ AccumT IlpReduce(int shift, T* data, int size, const Reduction<T, AccumT>& r, AccumT defaultVal) {
  using LoadT = aligned_vector<T, ILP>;
  AccumT threadVal = defaultVal;
  int offset = threadIdx.x;
  // shift and do 1
  if (shift > 0) {
    data -= shift;
    size += shift;
    if (threadIdx.x >= shift && threadIdx.x < size) {
      threadVal = r(threadVal, data[offset]);
    }
    size -= blockDim.x;
    data += blockDim.x;
  }
  if (size <= 0) return threadVal;
  int last = size % (ILP * blockDim.x);
  T v[ILP];
  LoadT* value = reinterpret_cast<LoadT*>(&v);
  for (; offset * ILP < (size - last); offset += blockDim.x) {
    *value = reinterpret_cast<LoadT*>(data)[offset];
#pragma unroll
    for (int j = 0; j < ILP; ++j) {
      threadVal = r(threadVal, v[j]);
    }
  }
  offset = size - last + threadIdx.x;
  // Epilogue
  for (; offset < size; offset += blockDim.x)
    threadVal = r(threadVal, data[offset]);
  return threadVal;
}

template <template <typename> class Reduction, typename AccumT>
__device__ __forceinline__ AccumT SoftmaxReduce(AccumT* smem, AccumT val, const Reduction<AccumT>& r, AccumT defaultVal) {
  // To avoid RaW races from chaining SoftmaxReduce calls together, we need a sync here
  __syncthreads();
  smem[threadIdx.x] = val;
  __syncthreads();
  AccumT warpVal = defaultVal;
  // First warp will perform per-warp reductions for the remaining warps
  if (threadIdx.x < kGPUWarpSize) {
    int warps_per_block = blockDim.x / kGPUWarpSize;
    for (int i = 0; i < warps_per_block; ++i) {
      warpVal = r(warpVal, smem[i * kGPUWarpSize + threadIdx.x]);
    }
    smem[threadIdx.x] = warpVal;
  }
  __syncthreads();
  // First thread will perform a reduction of the above per-warp reductions
  AccumT blockVal = defaultVal;
  if (threadIdx.x == 0) {
#pragma unroll
    for (int i = 0; i < kGPUWarpSize; ++i) {
      blockVal = r(blockVal, smem[i]);
    }
    smem[0] = blockVal;
  }
  // Sync and broadcast
  __syncthreads();
  return smem[0];
}

dim3 SoftmaxGetBlockSize(int ILP, uint64_t size) {
  uint64_t block_size = 1;
  uint64_t max_block_size = min(size / ILP, static_cast<uint64_t>(kMaxThreads));
  // In the vectorized case we want to trade off allowing more of the buffers to be accessed
  // in a vectorized way against wanting a larger block size to get better utilisation.
  // In general with ILP you can have (ILP-1)/ILP of the buffer accessed vectorised, at the risk
  // of having a very small block size. We choose to keep >= 1/2 of the buffer vectorised while
  // allowing a larger block size.
  if (ILP > 1) {
    max_block_size /= 2;
  }
  while (block_size < max_block_size) block_size *= 2;
  // Launch at least a single warp - the kernel assumes that.
  block_size = max(block_size, static_cast<uint64_t>(kGPUWarpSize));
  return dim3(static_cast<unsigned int>(block_size));
}

template <typename T, typename AccumT, typename OutT>
struct LogSoftmaxForwardEpilogue {
  __device__ __forceinline__ LogSoftmaxForwardEpilogue(AccumT max_input, AccumT sum)
      : max_input(max_input), logsum(log(sum)) {}

  __device__ __forceinline__ OutT operator()(T input) const {
    return static_cast<OutT>((AccumT)input - max_input - logsum);
  }

  const AccumT max_input;
  const AccumT logsum;
};

template <typename T, typename AccumT, typename OutT>
struct SoftmaxForwardEpilogue {
  __device__ __forceinline__ SoftmaxForwardEpilogue(AccumT max_input, AccumT sum)
      : max_input(max_input), sum(sum) {}

  __device__ __forceinline__ OutT operator()(T input) const {
    return static_cast<OutT>(exp((AccumT)input - max_input) / sum);
  }

  const AccumT max_input;
  const AccumT sum;
};

/**
 * This will apply the Epilogue with vectorized reads & writes when input & output have the same shift
 */
template <int ILP, typename scalar_t, typename accum_t, typename outscalar_t, template <typename, typename, typename> class Epilogue>
__device__ __forceinline__ void WriteFpropResultsVectorized(int size,
                                                            const int shift,
                                                            scalar_t* input,
                                                            outscalar_t* output,
                                                            Epilogue<scalar_t, accum_t, outscalar_t> epilogue) {
  using LoadT = aligned_vector<scalar_t, ILP>;
  using StoreT = aligned_vector<outscalar_t, ILP>;
  int offset = threadIdx.x;
  // if unaligned, do one value / thread and move on, guaranteeing aligned reads/writes later
  if (shift > 0) {
    input -= shift;
    output -= shift;
    size += shift;
    if (threadIdx.x >= shift && threadIdx.x < size) {
      output[offset] = epilogue(input[offset]);
    }
    size -= blockDim.x;
    input += blockDim.x;
    output += blockDim.x;
  }
  if (size <= 0) return;
  const int last = size % (ILP * blockDim.x);
  scalar_t in_v[ILP];
  LoadT* in_value = reinterpret_cast<LoadT*>(&in_v);
  outscalar_t out_v[ILP];
  StoreT* out_value = reinterpret_cast<StoreT*>(&out_v);
  for (; offset * ILP < (size - last); offset += blockDim.x) {
    *in_value = reinterpret_cast<LoadT*>(input)[offset];
#pragma unroll
    for (int j = 0; j < ILP; ++j) {
      out_v[j] = epilogue(in_v[j]);
    }
    reinterpret_cast<StoreT*>(output)[offset] = *out_value;
  }
  offset = size - last + threadIdx.x;
  // handle the tail
  for (; offset < size; offset += blockDim.x) {
    output[offset] = epilogue(input[offset]);
  }
}

/**
 * This will apply the Epilogue with non-vectrorized reads & writes for the general case
 */
template <int ILP, typename scalar_t, typename accum_t, typename outscalar_t, template <typename, typename, typename> class Epilogue>
__device__ __forceinline__ void WriteFpropResults(int classes,
                                                  scalar_t* input,
                                                  outscalar_t* output,
                                                  Epilogue<scalar_t, accum_t, outscalar_t> epilogue) {
  int offset = threadIdx.x;
  int last = classes % (ILP * blockDim.x);
  // Main bulk of loop with ILP
  for (; offset < classes - last; offset += blockDim.x * ILP) {
    scalar_t tmp[ILP];
#pragma unroll
    for (int j = 0; j < ILP; ++j) {
      tmp[j] = input[offset + j * blockDim.x];
    }
#pragma unroll
    for (int j = 0; j < ILP; ++j) {
      output[offset + j * blockDim.x] = epilogue(tmp[j]);
    }
  }
  // Remainder - no ILP
  for (; offset < classes; offset += blockDim.x) {
    output[offset] = epilogue(input[offset]);
  }
}

template <int ILP, typename scalar_t, typename accscalar_t, typename outscalar_t,
          template <typename, typename, typename> class Epilogue>
__global__ void SoftmaxBlockForward(outscalar_t* output, scalar_t* input, int classes,
                                    int input_stride, int output_stride) {
  extern __shared__ unsigned char smem[];
  auto sdata = reinterpret_cast<accscalar_t*>(smem);
  // forward pointers to batch[blockIdx.x]
  // each block handles a sample in the mini-batch
  input += blockIdx.x * input_stride;
  output += blockIdx.x * output_stride;
  const int input_align_bytes = ILP * sizeof(scalar_t);
  const int output_align_bytes = ILP * sizeof(outscalar_t);
  const int shift = ((uint64_t)input) % input_align_bytes / sizeof(scalar_t);
  const int output_shift = ((uint64_t)output) % output_align_bytes / sizeof(outscalar_t);
  // find the max
  accscalar_t threadMax = IlpReduce<MaxFloat, ILP, scalar_t, accscalar_t>(
      shift, input, classes, MaxFloat<scalar_t, accscalar_t>(), -std::numeric_limits<accscalar_t>::max());
  accscalar_t max_k = SoftmaxReduce<Max, accscalar_t>(
      sdata, threadMax, Max<accscalar_t>(), -std::numeric_limits<accscalar_t>::max());
  // reduce all values
  accscalar_t threadExp = IlpReduce<SumExpFloat, ILP, scalar_t, accscalar_t>(
      shift, input, classes, SumExpFloat<scalar_t, accscalar_t>(max_k), static_cast<accscalar_t>(0));
  accscalar_t sumAll = SoftmaxReduce<Add, accscalar_t>(
      sdata, threadExp, Add<accscalar_t>(), static_cast<accscalar_t>(0));
  Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(max_k, sumAll);
  if (shift == output_shift) {
    WriteFpropResultsVectorized<ILP, scalar_t, accscalar_t, outscalar_t, Epilogue>(classes, shift, input, output, epilogue);
  } else {
    WriteFpropResults<ILP, scalar_t, accscalar_t, outscalar_t, Epilogue>(classes, input, output, epilogue);
  }
}

template <bool is_log_softmax>
void DispatchBlockwiseSoftmaxForward(hipStream_t stream, float* output, const float* input, int softmax_elements,
                                     int input_stride, int output_stride, int batch_count) {
  dim3 grid(batch_count);
  constexpr int ILP = sizeof(float4) / sizeof(float);
  dim3 block = SoftmaxGetBlockSize(ILP, softmax_elements);
  if (is_log_softmax) {
    SoftmaxBlockForward<ILP, float, float, float, LogSoftmaxForwardEpilogue>
        <<<grid, block, block.x * sizeof(float), stream>>>(output, const_cast<float*>(input),
                                                           softmax_elements, input_stride, output_stride);
  } else {
    SoftmaxBlockForward<ILP, float, float, float, SoftmaxForwardEpilogue>
        <<<grid, block, block.x * sizeof(float), stream>>>(output, const_cast<float*>(input),
                                                           softmax_elements, input_stride, output_stride);
  }
  CUDA_CHECK(hipGetLastError());
}
template void DispatchBlockwiseSoftmaxForward<true>(hipStream_t, float*, const float*, int, int, int, int);
template void DispatchBlockwiseSoftmaxForward<false>(hipStream_t, float*, const float*, int, int, int, int);

// --- Sampling Kernels ---
template <int kBlockSize>
__global__ void PrefixSumKernel(float* scores, float* prefix_sums, int sample_range, int batch_size) {
  int batch = blockIdx.x;
  float prefix_sum = 0.0f;

  typedef hipcub::BlockScan<float, kBlockSize> BlockScan;
  __shared__ typename BlockScan::TempStorage temp_storage;

  for (int i = 0; i < sample_range; i += blockDim.x) {
    int global_index = threadIdx.x + i + batch * sample_range;
    int local_index = threadIdx.x + i;
    float score = (local_index < sample_range) ? scores[global_index] : 0.0f;
    float sum = score;
    BlockScan(temp_storage).InclusiveSum(sum, sum);
    prefix_sum += sum;
    __syncthreads();
    if (local_index < sample_range) {
      prefix_sums[global_index] = prefix_sum;
    }
  }
}

template <int kBlockSize>
__global__ void FilterOnTopP(float* scores, float* prefix_sums, float* scores_temp, float* actual_values, int sample_range, int batch_size, float p) {
  int batch = blockIdx.x;
  float prefix_sum = 0.0f;
  float saferNegative = std::numeric_limits<float>::lowest() / 1000.0f;

  typedef hipcub::BlockScan<float, kBlockSize> BlockScan;
  __shared__ typename BlockScan::TempStorage temp_storage;

  for (int i = 0; i < sample_range; i += blockDim.x) {
    int global_index = threadIdx.x + i + batch * sample_range;
    int local_index = threadIdx.x + i;
    float score = (local_index < sample_range) ? scores[global_index] : 0.0f;
    float sum = score;
    BlockScan(temp_storage).InclusiveSum(sum, sum);
    prefix_sum += sum;
    __syncthreads();
    if (local_index < sample_range) {
      scores_temp[global_index] = prefix_sum;
    }
    __syncthreads();
    if (local_index == 0) {
      prefix_sums[global_index] = actual_values[global_index];
    } else if (local_index < sample_range) {
      if (scores_temp[global_index - 1] < p) {
        prefix_sums[global_index] = actual_values[global_index];
      } else {
        prefix_sums[global_index] = saferNegative;
      }
    }
  }
}

__global__ void RandomThresholdKernel(hiprandState* curand_states, float* thresholds, int batch_size) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < batch_size) {
    // For numerical stability, we use 0.9999999f not 1.0f to avoid zero probabilities.
    thresholds[index] = 0.9999999f * hiprand_uniform(&curand_states[index]);
  }
}

template <int kBlockSize>
__global__ void SampleKernel(float* prefix_sums, int* indices, int* index_out, int sample_range, int indices_stride, float* thresholds) {
  int batch = blockIdx.x;
  int index = threadIdx.x;

  __shared__ int first_index;
  if (threadIdx.x == 0) {
    first_index = sample_range - 1;
  }
  __syncthreads();

  for (; index < sample_range - 1; index += blockDim.x) {
    float sum = prefix_sums[batch * sample_range + index];
    // TOP P or K
    if (sum >= thresholds[batch]) {
      atomicMin(&first_index, index);
      break;
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    index_out[batch] = indices[batch * indices_stride + first_index];
  }
}

void LaunchSampleKernel(SamplingData* data, hipStream_t stream, float* scores, int* indices, int* index_out, int sample_range, int batch_size, int indices_stride, float p, int k, float temperature) {
  dim3 grid(batch_size, 1, 1);
  dim3 block(256, 1, 1);

  // The `FilterOnTopP` kernel reads from `scores` (which contains probabilities) and writes the filtered results to `prefix_sums`.
  // Values that do not meet the Top-P criteria are set to a large negative number.
  FilterOnTopP<256><<<grid, block, 0, stream>>>(scores, data->prefix_sums.get(), data->scores_temp.get(), data->scores_buffer.get(), sample_range, batch_size, p);
  CUDA_CHECK(hipGetLastError());

  // After Top-P filtering, the remaining probabilities must be re-normalized.
  DispatchBlockwiseSoftmaxForward<false>(stream, data->scores_adjusted.get(), const_cast<const float*>(data->prefix_sums.get()), k, indices_stride, k, batch_size);
  CUDA_CHECK(hipGetLastError());

  PrefixSumKernel<256><<<grid, block, 0, stream>>>(data->scores_adjusted.get(), data->prefix_sums_adjusted.get(), sample_range, batch_size);
  CUDA_CHECK(hipGetLastError());

  RandomThresholdKernel<<<int((batch_size + 127) / 128), 128, 0, stream>>>(data->curand_states.get(), data->thresholds.get(), batch_size);
  CUDA_CHECK(hipGetLastError());

  SampleKernel<256><<<grid, block, 0, stream>>>(data->prefix_sums_adjusted.get(), indices, index_out, sample_range, indices_stride, data->thresholds.get());
  CUDA_CHECK(hipGetLastError());
}

// Main sampling entry point
void GetSample(SamplingData* data, hipStream_t stream, int32_t* next_token_out, float* scores_in, int vocab_size, int batch_size, int k, float p, float temperature) {
  if (k <= 0 || k > vocab_size) {
    k = vocab_size;
  }

  // Stage 1: Get Top K candidates.
  TopkData* topk_data = data;
  GetTopKSubset(topk_data, stream, scores_in, data->scores_sorted.get(), data->indices_sorted.get(), vocab_size, batch_size, k, temperature);

  // Stage 2: Sample from the top k candidates.
  int sample_range = k;
  int indices_stride = k;
  LaunchSampleKernel(data, stream, data->scores_sorted.get(), data->indices_sorted.get(), next_token_out, sample_range, batch_size, indices_stride, p, k, temperature);
}
}  // namespace cuda
}  // namespace Generators
